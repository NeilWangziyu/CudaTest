#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <conio.h>
#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

typedef unsigned int u32;
#define NUM_ELEM 32

__host__ void cpu_sort(u32 * const data, const u32 num_elements)
{
	static u32 cpu_tmp_0[NUM_ELEM];
	static u32 cpu_tmp_1[NUM_ELEM];

	for (u32 bit = 0; bit < 32; bit++)
	{
		u32 base_cnt_0 = 0;
		u32 base_cnt_1 = 0;

		for (u32 i = 0; i < num_elements; i++)
		{
			const u32 d = data[i];
			const u32 bit_mask = (1 << bit);

			if ((d & bit_mask) > 0)
			{
				cpu_tmp_1[base_cnt_1] = d;
				base_cnt_1++;
			}
			else
			{
				cpu_tmp_0[base_cnt_0] = d;
				base_cnt_0++;
			}

		}

		//copy data back to source - first the zero first
		for (u32 i = 0; i < base_cnt_0; i++)
		{
			data[i] = cpu_tmp_0[i];
		}

		//copy data back to source - then the one list
		for (u32 i = 0; i < base_cnt_1; i++)
		{
			data[base_cnt_0 + i] = cpu_tmp_1[i];
		}

	}

}

__device__ void radix_sort(u32 * const sort_tmp,
	const u32 num_lists,
	const u32 num_elements,
	const u32 tid,
	u32 * const sort_tmp_0,
	u32 * const sort_tmp_1)
{
	//sort into num_list, lists
	// Apply radix sort on 32 bits of data
	for (u32 bit = 0; bit < 32; bit++)
	{
		u32 base_cnt_0 = 0;
		u32 base_cnt_1 = 0;

		for (u32 i = 0; i < num_elements; i+=num_lists)
		{
			const u32 elem = sort_tmp[i + tid];
			const u32 bit_mask = (1 << bit);

			if ((elem & bit_mask) > 0)
			{
				sort_tmp_1[base_cnt_1 + tid] = elem;
				base_cnt_1 += num_lists;
			}
			else
			{
				sort_tmp_0[base_cnt_0 + tid] = elem;
				base_cnt_0 += num_lists;
			}
		}

		//copy data back to source - first the zero list
		for (u32 i = 0; i < base_cnt_0; i+=num_lists)
		{
			sort_tmp[i + tid] = sort_tmp_0[i + tid];
		}

		for (u32 i = 0; i < base_cnt_1; i += num_lists)
		{
			sort_tmp[base_cnt_0 + i + tid] = sort_tmp_1[i + tid];
		}

	}

	__syncthreads();
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	printf("CPU version of bin sort.\n");

	u32 input[arraySize] = { 11, 2, 9, 4, 5 };

	printf("Before sort: {%d,%d,%d,%d,%d}\n",
		input[0], input[1], input[2], input[3], input[4]);

	cpu_sort(input, arraySize);

	printf("After sort: {%d,%d,%d,%d,%d}\n",
		input[0], input[1], input[2], input[3], input[4]);


    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
