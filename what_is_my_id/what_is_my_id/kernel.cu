#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <conio.h>


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}



__global__ void what_is_my_id(unsigned int* const block, 
			unsigned int* const thread,
			unsigned int* const warp,
			unsigned int* const calc_thread)
{
	//thread id is block index * block size, size + thread offset into the block
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	block[thread_idx] = blockIdx.x;
	thread[thread_idx] = threadIdx.x;

	//calculate waro using build in variable warpsize
	warp[thread_idx] = threadIdx.x / warpSize;

	calc_thread[thread_idx] = thread_idx;

}

#define ARRAY_SIZE 128
#define ARRAY_SIZE_IN_BYTES (sizeof(unsigned int) * (ARRAY_SIZE))

//declare statically four array of ARRAY_SIZE each

unsigned int cpu_block[ARRAY_SIZE];
unsigned int cpu_thread[ARRAY_SIZE];
unsigned int cpu_warp[ARRAY_SIZE];
unsigned int cpu_calc_thread[ARRAY_SIZE];


int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

     /*hipDeviceReset must be called before exiting in order for profiling and
     tracing tools such as Nsight and Visual Profiler to show complete traces.*/

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }


	printf("default is finished ! and now we running the what_is_my_id");


	//total thread count = 2 * 64 = 128
	const unsigned int num_blocks = 2;
	const unsigned int num_threads = 64;
	char ch;

	//declare pointers for GPU based params
	unsigned int* gpu_block;
	unsigned int* gpu_thread;
	unsigned int* gpu_warp;
	unsigned int* gpu_calc_thread;

	//decalre loop counter for use later
	unsigned int i;

	//allocate four arays on the GPU
	hipMalloc((void **)&gpu_block, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_thread, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_warp, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_calc_thread, ARRAY_SIZE_IN_BYTES);


	//execute our kernel
	what_is_my_id<<<num_blocks, num_threads>>>(gpu_block, gpu_thread, gpu_warp, gpu_calc_thread);

	//copy back the gpu resylt to the GPU
	hipMemcpy(cpu_block, gpu_block, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_thread, gpu_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_warp, gpu_warp, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
	hipMemcpy(cpu_calc_thread, gpu_calc_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);

	//free the array on the GPU as now we are done with them
	hipFree(gpu_block);
	hipFree(gpu_thread);
	hipFree(gpu_warp);
	hipFree(gpu_calc_thread);

	for (i = 0; i < ARRAY_SIZE; i++)
	{
		printf("Calculated Thread: %3u - Block: %2u - Warp %2u - Thread %3u\n",
			cpu_calc_thread[i], cpu_block[i], cpu_warp[i], cpu_thread[i]);
	}

	ch = getch();


    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
