#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <conio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}


__global__ void what_is_my_id_2d_A(
	unsigned int * const block_x,
	unsigned int * const block_y,
	unsigned int * const thread,
	unsigned int * const calc_thread,
	unsigned int * const x_thread,
	unsigned int * const y_thread,
	unsigned int * const grid_dimx,
	unsigned int * const block_dimx,
	unsigned int * const grid_dimy,
	unsigned int * const block_dimy
	)
{
	const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	const unsigned int idy = (blockIdx.y * blockDim.y) + threadIdx.y;

	const unsigned int thread_idx = ((gridDim.x * gridDim.x) * idy) + idx;

	block_x[thread_idx] = blockIdx.x;
	block_y[thread_idx] = blockIdx.y;
	thread[thread_idx] = threadIdx.x;
	calc_thread[thread_idx] = thread_idx;
	x_thread[thread_idx] = idx;
	y_thread[thread_idx] = idy;
	grid_dimx[thread_idx] = gridDim.x;
	block_dimx[thread_idx] = blockDim.x;
	grid_dimy[thread_idx] = gridDim.y;
	block_dimy[thread_idx] = blockDim.y;


}

#define ARRAY_SIZE_X 32
#define ARRAY_SIZE_Y 16

#define ARRAY_SIZE_IN_BYTES ((ARRAY_SIZE_X) * (ARRAY_SIZE_Y) * (sizeof(unsigned int)))

//declare statically six arrary of array_size each
unsigned int cpu_block_x[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_y[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_thread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_warp[ARRAY_SIZE_Y][ARRAY_SIZE_X];

unsigned int cpu_calc_thread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_xthread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_ythread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_grid_dimx[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_dimx[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_grid_dimy[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_dimy[ARRAY_SIZE_Y][ARRAY_SIZE_X];



int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	// below is our code begins

	//total thread count = 32 * 4
	const dim3 threads_rect(32, 4);
	const dim3 blocks_rect(1, 4);

	//total thread count 16 * 8 = 128

	const dim3 threads_square(16, 8);
	const dim3 blocks_square(2, 2);

	//needed to wait for a character to exit
	char ch;

	//decalre pointers for GPU based params
	unsigned int * gpu_block_x;
	unsigned int * gpu_block_y;

	unsigned int * gpu_thread;
	unsigned int * gpu_warp;
	unsigned int * gpu_calc_thread;
	unsigned int * gpu_xthread;
	unsigned int * gpu_ythread;

	unsigned int * gpu_grid_dimx;
	unsigned int * gpu_block_dimx;
	unsigned int * gpu_grid_dimy;
	unsigned int * gpu_block_dimy;


	//allocate fourt arrarys on the GPU
	hipMalloc((void **)&gpu_block_x, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_block_y, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_thread, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_calc_thread, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_xthread, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_ythread, ARRAY_SIZE_IN_BYTES);

	hipMalloc((void **)&gpu_grid_dimx, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_block_dimx, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_grid_dimy, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void **)&gpu_block_dimy, ARRAY_SIZE_IN_BYTES);

	for (int kernel = 0; kernel < 2; kernel++)
	{
		switch (kernel)
		{
		case 0:
		{
			//execute our kernel
			what_is_my_id_2d_A <<<blocks_square, threads_rect>>> (gpu_block_x, gpu_block_y, gpu_thread, gpu_calc_thread,
				gpu_xthread, gpu_ythread, gpu_grid_dimx, gpu_block_dimx, gpu_grid_dimy, gpu_block_dimy);
		}break;
		case 1:
		{
			what_is_my_id_2d_A <<<blocks_square, threads_square>>> (gpu_block_x, gpu_block_y, gpu_thread, gpu_calc_thread,
				gpu_xthread, gpu_ythread, gpu_grid_dimx, gpu_block_dimx, gpu_grid_dimy, gpu_block_dimy);

		}break;
		default: exit(1); break;
		}

		hipMemcpy(cpu_block_x, gpu_block_x, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
		hipMemcpy(cpu_block_y, gpu_block_y, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
		hipMemcpy(cpu_thread, gpu_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
		hipMemcpy(cpu_calc_thread, gpu_calc_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);

		hipMemcpy(cpu_xthread, gpu_xthread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
		hipMemcpy(cpu_xthread, gpu_xthread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);

		hipMemcpy(cpu_grid_dimx, gpu_grid_dimx, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
		hipMemcpy(cpu_block_dimx, gpu_block_dimx, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
		hipMemcpy(cpu_grid_dimy, cpu_grid_dimy, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
		hipMemcpy(cpu_block_dimy, gpu_block_dimy, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);


		printf("\nKernal %d\n", kernel);

		//iterate through the arrays and print
		for (int y = 0; y < ARRAY_SIZE_Y; y++)
		{
			for (int x = 0; x < ARRAY_SIZE_X; x++)
			{
				printf("CT:%2u  BKX:%1u  TID:%2u  YTID:%2u  XTID:%2u  GDX:%1u  BDX:%1u  GDY %1u  BDY %1u\n",
					cpu_calc_thread[y][x], cpu_block_x[y][x], cpu_block_y[y][x], cpu_thread[y][x], cpu_ythread[y][x], cpu_xthread[y][x],
					cpu_grid_dimx[y][x], cpu_block_dimx[y][x], cpu_grid_dimy[y][x], cpu_block_dimy[y][x]);

				ch = getch();
			}

		}

		printf("PRESS ABT KEY TO CONTINUE\n");
		ch = getch();

	}

	hipFree(gpu_block_x);
	hipFree(gpu_block_y);
	hipFree(gpu_thread);
	hipFree(gpu_calc_thread);

	hipFree(gpu_xthread);
	hipFree(gpu_ythread);

	hipFree(gpu_grid_dimx);
	hipFree(gpu_block_dimx);
	hipFree(gpu_grid_dimy);
	hipFree(gpu_block_dimy);


    return 0;

	// it looks like something is wrong in the code
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
